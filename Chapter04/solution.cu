#include "hip/hip_runtime.h"
// host call global
// global call device

#include <stdio.h>

__device__ void Device1()
{
    printf("Device1\n");
}

__device__ void Device2()
{
    printf("Device2");
}

__global__ void kernel()
{
    Device1();
    Device2();
}

void sub_Function_in_Host()
{
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}

int main()
{
    sub_Function_in_Host();
    return 0;
}

// Device1
// Device2

//-----------------------------------------------------------------------
// host call device
#include <stdio.h>

__device__ void Device1()
{
    printf("Device1\n");
}

__device__ void Device2()
{
    printf("Device2");
}

void sub_Function_in_Host()
{
    Device1();
}

int main()
{
    sub_Function_in_Host();
    Device2();
    hipDeviceSynchronize();
    return 0;
}

// error: calling a __device__ function("Device1") from a __host__ function("sub_Function_in_Host") is not allowed
// error: calling a __device__ function("Device2") from a __host__ function("main") is not allowed

//---------------------------------------------------------------------
// device call host
#include <stdio.h>

void sub_Function_in_Host()
{
    printf("host function");
}

__device__ void Device1()
{
    sub_Function_in_Host();
}

int main()
{
    Device1();
    hipDeviceSynchronize();
    return 0;
}

// error: calling a __host__ function("sub_Function_in_Host") from a __device__ function("Device1") is not allowed
// error: identifier "sub_Function_in_Host" is undefined in device code

//------------------------------------------------------------
// global call host

#include <stdio.h>

void sub_Function_in_Host()
{
    printf("host function");
}

__global__ void kernel()
{
    sub_Function_in_Host();
}

int main()
{
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}

// error: calling a __host__ function("sub_Function_in_Host") from a __global__ function("kernel") is not allowed
// error: identifier "sub_Function_in_Host" is undefined in device code

//-----------------------------------------------------------------
// device call global

#include <stdio.h>

__global__ void kernel()
{
    printf("kernel function");
}

__device__ void Device1()
{
    kernel<<<1, 1>>>();
}

int main()
{
    Device1();
    hipDeviceSynchronize();
    return 0;
}

// error: calling a __global__ function("kernel") from a __device__ function("Device1") is only allowed on the compute_35 architecture or above

// -----------------------------------------------------------
#include <stdio.h>

__global__ void kernel1()
{
    printf("kernel1\n");
}

__global__ void kernel2()
{
    printf("kernel2\n");
}

int main()
{
    kernel1<<<1, 1>>>();
    printf("CPU here\n");
    kernel2<<<1, 1>>>();
    hipDeviceSynchronize();
    printf("CPU also here\n");
    return 0;
}

// CPU here
// kernel1
// kernel2
// CPU also here
