
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 10
#define NUM_BLOCKS 2
#define ARRAY_SIZE 20 

__global__ void AtomicAdd(int *result, int *array_add)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    atomicAdd(result, array_add[tid]);

    // if (threadIdx.x == 0)
    // {
    //     atomicAdd(result, array_add[tid]);
    // }
}

__global__ void AtomicSub(int *result, int *array_sub)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    atomicSub(result, array_sub[tid]);

    // if (threadIdx.x == 0)
    // {
    //     atomicSub(result, array_sub[tid]);
    // }
}

__global__ void AtomicMax(int *result, int *array_max)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    atomicMax(result, array_max[tid]);

    // if (threadIdx.x == 0)
    // {
    //     atomicMax(result, array_max[tid]);
    // }
}

__global__ void AtomicMin(int *result, int *array_min)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    atomicMin(result, array_min[tid]);

    // if (threadIdx.x == 0)
    // {
    //     atomicMin(result, array_min[tid]);
    // }
}

int main()
{
    int *h_data = (int *)malloc(ARRAY_SIZE * sizeof(int));
    int *d_data;
    hipMalloc((void **)&d_data, ARRAY_SIZE * sizeof(int));

    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        h_data[i] = i;
    }

    //------------ atomicAdd-------------
    int *d_result_add;
    hipMalloc((void **)&d_result_add, sizeof(int));
    int h_result_add = 0;

    //------------ atomicSub-------------
    int *d_result_sub;
    hipMalloc((void **)&d_result_sub, sizeof(int));
    int h_result_sub = 0;

    //------------ atomicMax-------------
    int *d_result_max;
    hipMalloc((void **)&d_result_max, sizeof(int));
    int h_result_max = 0;

    //------------ atomicMin-------------
    int *d_result_min;
    hipMalloc((void **)&d_result_min, sizeof(int));
    int h_result_min = 0;

    hipMemcpy(d_data, h_data, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);

    AtomicAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_result_add, d_data);
    AtomicSub<<<NUM_BLOCKS, NUM_THREADS>>>(d_result_sub, d_data);
    AtomicMax<<<NUM_BLOCKS, NUM_THREADS>>>(d_result_max, d_data);
    AtomicMin<<<NUM_BLOCKS, NUM_THREADS>>>(d_result_min, d_data);

    hipMemcpy(&h_result_add, d_result_add, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_result_sub, d_result_sub, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_result_max, d_result_max, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_result_min, d_result_min, sizeof(int), hipMemcpyDeviceToHost);

    printf("Atomic Add Result: %d\n", h_result_add);
    printf("Atomic Sub Result: %d\n", h_result_sub);
    printf("Atomic Max Result: %d\n", h_result_max);
    printf("Atomic Min Result: %d\n", h_result_min);

    free(h_data);

    hipFree(d_result_add);
    hipFree(d_result_sub);
    hipFree(d_result_max);
    hipFree(d_result_min);
    hipFree(d_data);

    return 0;
}
