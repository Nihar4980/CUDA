#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE (16 * 1024 * 1024)



void pageableMemoryTest() {
    float *h_data, *d_data;
    h_data = (float *)malloc(SIZE * sizeof(float));
    hipMalloc((void **)&d_data, SIZE * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Host to Device
    hipEventRecord(start);
    hipMemcpy(d_data, h_data, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Pageable - Host to Device: %f ms\n", milliseconds);

    // Device to Host
    hipEventRecord(start);
    hipMemcpy(h_data, d_data, SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Pageable - Device to Host: %f ms\n", milliseconds);

    free(h_data);
    hipFree(d_data);
}

void pinnedMemoryTest() {
    float *h_data, *d_data;
    hipHostMalloc((void **)&h_data, SIZE * sizeof(float), hipHostMallocDefault);
    hipMalloc((void **)&d_data, SIZE * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Host to Device
    hipEventRecord(start);
    hipMemcpy(d_data, h_data, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Pinned - Host to Device: %f ms\n", milliseconds);

    // Device to Host
    hipEventRecord(start);
    hipMemcpy(h_data, d_data, SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Pinned - Device to Host: %f ms\n", milliseconds);

    hipHostFree(h_data);
    hipFree(d_data);
}

int main() {
    printf("Running pageable memory test...\n");
    pageableMemoryTest();

    printf("\nRunning pinned memory test...\n");
    pinnedMemoryTest();

    return 0;
}

